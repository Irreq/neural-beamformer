#include "hip/hip_runtime.h"
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include "../config.h"


__global__ void miso(const float *in, const float *h, float *out)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    int offset;

    if (i < N_SAMPLES - 1)
    {
        out[i] = 0.f;
        for (int k = 0; k < N_SENSORS; k++)
        {
            offset = k * N_SAMPLES + i;
            out[i] += in[offset] + h[k] * (in[offset] - in[offset + 1]);
        }
    }
}


void main()
{
    size_t size = N_SAMPLES * N_SENSORS * sizeof(float);

    float *h_signal = (float *)malloc(size);

    float *h_delay = (float *)malloc(N_SENSORS * sizeof(float));

    float *h_result = (float *)malloc(N_SAMPLES * sizeof(float));

    if (h_signal == NULL || h_delay == NULL || h_result == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    for (int k = 0; k < N_SENSORS; k++)
    {
        h_delay[k] = 0.5f; 
        for (int i = 0; i < N_SAMPLES; i++)
        {
            result[i] = 0.f;
            h_signal[k * N_SAMPLES + i] = (float)i;
        }
    }

    printf("All worked well!\n");


    free(h_signal);
    free(h_delay);
    free(h_result);
}
